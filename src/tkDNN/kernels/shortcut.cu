#include "hip/hip_runtime.h"
#include "kernels.h"
#include "assert.h"

__global__ void shortcut_kernel(int size, int minw, int minh, int minc, int stride, int sample, int batch, 
                                int w1, int h1, int c1, dnnType *add, 
                                int w2, int h2, int c2, float s1, float s2, dnnType *out)
{
    int id = (blockIdx.x + blockIdx.y*gridDim.x) * blockDim.x + threadIdx.x;
    if (id >= size) return;
    int i = id % minw;
    id /= minw;
    int j = id % minh;
    id /= minh;
    int k = id % minc;
    id /= minc;
    int b = id % batch;

    int out_index = i*sample + w2*(j*sample + h2*(k + c2*b));
    int add_index = i*stride + w1*(j*stride + h1*(k + c1*b));
    out[out_index] = s1*out[out_index] + s2*add[add_index];
    //out[out_index] += add[add_index];
}

void shortcutForward(dnnType* srcData, dnnType* dstData, int n1, int c1, int h1, int w1, int s1,
                                                         int n2, int c2, int h2, int w2, int s2, 
                     hipStream_t stream)
{
    assert(n1 == n2);
    int batch = n1;

    int minw = (w1 < w2) ? w1 : w2;
    int minh = (h1 < h2) ? h1 : h2;
    int minc = (c1 < c2) ? c1 : c2;

    int stride = w1/w2;
    int sample = w2/w1;
    assert(stride == h1/h2);
    assert(sample == h2/h1);
    if(stride < 1) stride = 1;
    if(sample < 1) sample = 1;

    int size = batch * minw * minh * minc;
    int blocks = (size+255)/256;
    int threads = 256;
    shortcut_kernel<<<blocks, threads, 0, stream>>>(size, minw, minh, minc, stride, sample, batch, 
        w1, h1, c1, srcData, w2, h2, c2, s1, s2, dstData);
}
