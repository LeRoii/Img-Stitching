#include "hip/hip_runtime.h"
#include "kernels.h"

__global__
void activation_logistic(dnnType *input, dnnType *output, int size) {

    int i = blockDim.x*blockIdx.x + threadIdx.x;

    if(i<size) {    
        output[i] =  1.0f/(1.0f + exp(-input[i]));;
    }
 }


/**
    LOGISTIC activation function
*/
void activationLOGISTICForward(dnnType* srcData, dnnType* dstData, int size, hipStream_t stream)
{
    int blocks = (size+255)/256;
    int threads = 256;
    
    activation_logistic<<<blocks, threads, 0, stream>>>(srcData, dstData, size);
}


