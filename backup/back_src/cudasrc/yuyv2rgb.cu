#include "hip/hip_runtime.h"
#include <sys/time.h>
#include "hip/hip_runtime.h"
#include "yuyv2rgb.cuh"

static __device__ const unsigned char uchar_clipping_table[] = {
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, // -128 - -121
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, // -120 - -113
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, // -112 - -105
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, // -104 -  -97
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -96 -  -89
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -88 -  -81
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -80 -  -73
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -72 -  -65
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -64 -  -57
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -56 -  -49
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -48 -  -41
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -40 -  -33
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -32 -  -25
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -24 -  -17
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //  -16 -   -9
	0,
	0,
	0,
	0,
	0,
	0,
	0,
	0, //   -8 -   -1
	0, 1, 2, 3, 4, 5, 6, 7, 8, 9, 10, 11, 12, 13, 14, 15, 16, 17, 18, 19, 20, 21, 22, 23, 24, 25, 26, 27, 28, 29, 30,
	31, 32, 33, 34, 35, 36, 37, 38, 39, 40, 41, 42, 43, 44, 45, 46, 47, 48, 49, 50, 51, 52, 53, 54, 55, 56, 57, 58, 59,
	60, 61, 62, 63, 64, 65, 66, 67, 68, 69, 70, 71, 72, 73, 74, 75, 76, 77, 78, 79, 80, 81, 82, 83, 84, 85, 86, 87, 88,
	89, 90, 91, 92, 93, 94, 95, 96, 97, 98, 99, 100, 101, 102, 103, 104, 105, 106, 107, 108, 109, 110, 111, 112, 113,
	114, 115, 116, 117, 118, 119, 120, 121, 122, 123, 124, 125, 126, 127, 128, 129, 130, 131, 132, 133, 134, 135, 136,
	137, 138, 139, 140, 141, 142, 143, 144, 145, 146, 147, 148, 149, 150, 151, 152, 153, 154, 155, 156, 157, 158, 159,
	160, 161, 162, 163, 164, 165, 166, 167, 168, 169, 170, 171, 172, 173, 174, 175, 176, 177, 178, 179, 180, 181, 182,
	183, 184, 185, 186, 187, 188, 189, 190, 191, 192, 193, 194, 195, 196, 197, 198, 199, 200, 201, 202, 203, 204, 205,
	206, 207, 208, 209, 210, 211, 212, 213, 214, 215, 216, 217, 218, 219, 220, 221, 222, 223, 224, 225, 226, 227, 228,
	229, 230, 231, 232, 233, 234, 235, 236, 237, 238, 239, 240, 241, 242, 243, 244, 245, 246, 247, 248, 249, 250, 251,
	252, 253, 254, 255, 255, 255, 255, 255, 255, 255, 255, 255, // 256-263
	255, 255, 255, 255, 255, 255, 255, 255, // 264-271
	255, 255, 255, 255, 255, 255, 255, 255, // 272-279
	255, 255, 255, 255, 255, 255, 255, 255, // 280-287
	255, 255, 255, 255, 255, 255, 255, 255, // 288-295
	255, 255, 255, 255, 255, 255, 255, 255, // 296-303
	255, 255, 255, 255, 255, 255, 255, 255, // 304-311
	255, 255, 255, 255, 255, 255, 255, 255, // 312-319
	255, 255, 255, 255, 255, 255, 255, 255, // 320-327
	255, 255, 255, 255, 255, 255, 255, 255, // 328-335
	255, 255, 255, 255, 255, 255, 255, 255, // 336-343
	255, 255, 255, 255, 255, 255, 255, 255, // 344-351
	255, 255, 255, 255, 255, 255, 255, 255, // 352-359
	255, 255, 255, 255, 255, 255, 255, 255, // 360-367
	255, 255, 255, 255, 255, 255, 255, 255, // 368-375
	255, 255, 255, 255, 255, 255, 255, 255, // 376-383
};

/** Clip a value to the range 0<val<255. For speed this is done using an
 * array, so can only cope with numbers in the range -128<val<383.
 */
static __device__ unsigned char CLIPVALUE(int val)
{
	// Old method (if)
	/* val = val < 0 ? 0 : val; */
	/* return val > 255 ? 255 : val; */
	
	// New method (array)
	const int clipping_table_offset = 128;
	return uchar_clipping_table[val + clipping_table_offset];
}

static __device__ void YUV2RGB(const unsigned char y, const unsigned char u, const unsigned char v, unsigned char* r,
                    unsigned char* g, unsigned char* b)
{
	const int y2 = (int)y;
	const int u2 = (int)u - 128;
	const int v2 = (int)v - 128;
	//std::cerr << "YUV=("<<y2<<","<<u2<<","<<v2<<")"<<std::endl;
	
	// This is the normal YUV conversion, but
	// appears to be incorrect for the firewire cameras
	/* int r2 = y2 + ( (v2*91947) >> 16); */
	/* int g2 = y2 - ( ((u2*22544) + (v2*46793)) >> 16 ); */
	/* int b2 = y2 + ( (u2*115999) >> 16); */

	// This is an adjusted version (UV spread out a bit)
	int r2 = y2 + ((v2 * 37221) >> 15);
	int g2 = y2 - (((u2 * 12975) + (v2 * 18949)) >> 15);
	int b2 = y2 + ((u2 * 66883) >> 15);
	//std::cerr << "   RGB=("<<r2<<","<<g2<<","<<b2<<")"<<std::endl;
	
	// Cap the values.
	*r = CLIPVALUE(r2);
	*g = CLIPVALUE(g2);
	*b = CLIPVALUE(b2);
}

__global__ void yuyv2rgb(char *YUV, char *RGB)
{
	unsigned char y0, y1, u, v;
	unsigned char r0, g0, b0;
	unsigned char r1, g1, b1;
	
	int nIn = blockIdx.x * blockDim.x * 4 + threadIdx.x * 4;
	y0 = (unsigned char)YUV[nIn];
	u  = (unsigned char)YUV[nIn + 1];
	y1 = (unsigned char)YUV[nIn + 2];
	v  = (unsigned char)YUV[nIn + 3];
	
	YUV2RGB(y0, u, v, &r0, &g0, &b0);
	YUV2RGB(y1, u, v, &r1, &g1, &b1);
	
	int nOut = blockIdx.x * blockDim.x * 6 + threadIdx.x * 6;
	RGB[nOut] = r0;
	RGB[nOut + 1] = g0;
	RGB[nOut + 2] = b0;
	RGB[nOut + 3] = r1;
	RGB[nOut + 4] = g1;
	RGB[nOut + 5] = b1;
}

void yuyv2rgb_cuda(char* YUV, char* RGB, int num_blocks, int block_size)
{
	yuyv2rgb<<<num_blocks, block_size>>>(YUV, RGB);
	hipDeviceSynchronize();
}
/*
extern "C" void process_image_cuda(const void *src, int size)
{
        struct timeval ts;

        int yuv_size = size * sizeof(char);

        gettimeofday(&ts, NULL);
        printf("[%lu.%lu]\tbefore copy image_data(CPU to GPU)\n", ts.tv_sec, ts.tv_usec);
        hipError_t ret = hipMemcpy(image_yuyv_cuda_, src, yuv_size, hipMemcpyHostToDevice);
        gettimeofday(&ts, NULL);
        printf("[%lu.%lu]\tcopy image_data(CPU to GPU) done\n", ts.tv_sec, ts.tv_usec);

        if (hipSuccess != ret) {
                printf("hipMemcpy fail %d\n", ret);
        }
        const int block_size = 256;
        const int num_blocks = yuv_size / (4*block_size);


        gettimeofday(&ts, NULL);
        printf("[%lu.%lu]\tbefore yuyv2rgb computation\n", ts.tv_sec, ts.tv_usec);
        yuyv2rgb_cuda(image_yuyv_cuda_, image_rgb_cuda_, num_blocks, block_size);
        gettimeofday(&ts, NULL);
        printf("[%lu.%lu]\tyuyv2rgb computation done\n", ts.tv_sec, ts.tv_usec);


        int rgb_size = size / 2 * 3 * sizeof(char);

        gettimeofday(&ts, NULL);
        printf("[%lu.%lu]\tbefore copy image_data(GPU to CPU)\n", ts.tv_sec, ts.tv_usec);
        ret = hipMemcpy(show_buf, image_rgb_cuda_, rgb_size, hipMemcpyDeviceToHost);
        gettimeofday(&ts, NULL);
        printf("[%lu.%lu]\tcopy image_data(GPU to CPU) done\n", ts.tv_sec, ts.tv_usec);
        printf("[%lu.%lu]\tcuda process image index = %d\n", ts.tv_sec, ts.tv_usec, ++index_pro);

        if (hipSuccess != ret) {
                printf("hipMemcpy fail %d\n", ret);
        }
}
*/
